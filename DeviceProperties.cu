#include<stdio.h>
#include<hip/hip_runtime.h>


int main(void) {

	// function declaration
	void printCudaDeviceProperties();

	// code
	printCudaDeviceProperties();


}

void printCudaDeviceProperties() {

	// code
	printf("CUDA Information...!!\n");

	printf("=====================\n");

	hipError_t ret_cuda_rt;

	int devCount;
	ret_cuda_rt = hipGetDeviceCount(&devCount);
	if (ret_cuda_rt != hipSuccess) {
		
		printf("Cuda Runtime API Error - cudaDeviceCount() failed.");
	}
	else if (devCount == 0) {
		printf("There is no cuda supported device on this system.");
		return;
	}
	else {
		printf("Total number of CUDA supporting GPU Device(s) on this system : %d\n", devCount);
		for (int i = 0; i < devCount; i++) {
			hipDeviceProp_t devProp;
			int driverVersion = 0, runtimeVersion = 0;
			ret_cuda_rt = hipGetDeviceProperties(&devProp, i);
			
			if (ret_cuda_rt != hipSuccess) {
			
				printf("%s in %s at line %d\n", hipGetErrorString(ret_cuda_rt), __FILE__, __LINE__);
			
				return;
			}
			printf("\n");
			hipDriverGetVersion(&driverVersion);
			hipRuntimeGetVersion(&runtimeVersion);

			printf("***** Cuda driver and Runtime info ****\n");
			printf("=======================================\n");
			printf("Cuda driver version   : %d.%d\n", driverVersion / 1000, (driverVersion % 100) / 10);
			printf("Cuda Runtime version  : %d.%d\n", runtimeVersion / 1000, (runtimeVersion % 100) / 10);
			printf("\n");
			printf("=======================================\n");
			printf("***** GPU General Information ****\n");
			printf("=======================================\n");
			printf("GPU Device Number : %d\n", i);
			printf("GPU Device Name :%s\n", devProp.name);
			printf("GPU Device Compute Capability :%d.%d\n", devProp.major, devProp.minor);
			printf("GPU Clock Rate : %d\n", devProp.clockRate);
			printf("GPU Device Type : ");
			if (devProp.integrated) 
				printf("Integrated (On-board) \n");
			else
				printf("Discrete (Card)\n");

			printf("\n");
			printf("=======================================\n");
			printf("***** GPU Device Memory Information ****\n");
			printf("=======================================\n");
			printf("GPU Device Total Memory GB = %.0f MB =%llu Bytes\n", ((float) devProp.totalGlobalMem / 1048576.0f) / 1024.0f, (unsigned long long) devProp.totalGlobalMem);
			printf("GPU Device Constant Memory   :%lu Bytes \n", (unsigned long)devProp.totalConstMem);
			printf("GPU Device Shared Memory Per SMProcessor     :%lu\n", (unsigned long)devProp.sharedMemPerBlock);
			printf("\n");
			printf("=======================================\n");
			printf("***** GPU Device Multiprocessor Information ****\n");
			printf("=======================================\n");
			printf("GPU Device Number of SMProcessors :%d\n", devProp.multiProcessorCount);
			printf("GPU Device Numner of Registers Per SMProcessor :%d\n", devProp.regsPerBlock);
			printf("=======================================\n");
			printf("***** GPU Device Thread Information ****\n");
			printf("=======================================\n");
			printf("GPU Device Maximum Number of Threads Per Block :%d\n", devProp.maxThreadsPerMultiProcessor);
			printf("GPU Device Maximum Threads in Warp		  : %d\n", devProp.warpSize);
			printf("GPU Device Maximum Thread Dimensions      : (%d, %d, %d )\n", devProp.maxThreadsDim[0], devProp.maxThreadsDim[1], devProp.maxThreadsDim[2]);
			printf("GPU Device Maximum Thread Grid Dimensions : (%d, %d, %d)\n", devProp.maxGridSize[0], devProp.maxGridSize[1], devProp.maxGridSize[2]);
			printf("\n");
			printf("=======================================\n");
			printf("***** GPU Device Driver Information ****\n");
			printf("=======================================\n");
			printf("GPU Device has ECC (Err. Correction Code) support :%s\n", devProp.ECCEnabled ? "Enabled" : "Disabled");
#if  defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
			printf("GPU Device CUDA Driver Mode (TCC or WDDM)  : %s\n", devProp.tccDriver ? "TCC (Tesla Compute Cluster Driver)" : "WDDM (Windows Display Driver Model)");
#endif
			printf("****************************************\n");
		
			}
		}
	}
